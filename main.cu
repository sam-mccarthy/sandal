#include <iostream>
#include <string>
#include "fpng.cuh"

int main() {
    auto byteSize = sizeof(float) * size2;
    float* theta1Ptr;
    float* theta2Ptr;
    float* velocity1Ptr;
    float* velocity2Ptr;

    hipMalloc((void **)&theta1Ptr, byteSize);
    hipMalloc((void **)&theta2Ptr, byteSize);
    hipMalloc((void **)&velocity1Ptr, byteSize);
    hipMalloc((void **)&velocity2Ptr, byteSize);
    hipMemcpy(theta1Ptr, theta1, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(theta2Ptr, theta2, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(theta1Ptr, theta1, byteSize, hipMemcpyHostToDevice);
    hipMemcpy(theta2Ptr, theta2, byteSize, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);

    auto imgSize = sizeof(unsigned char) * size2 * 4;
    auto* image = new unsigned char[imgSize];

    unsigned char* imagePtr;
    hipMalloc((void **)&imagePtr, imgSize);

    int i = 0;
    while(++i) {
        std::cout << "Calculating frame " << i;
        CalculateFrame<<<numBlocks, threadsPerBlock>>>(theta1Ptr, theta2Ptr, velocity1Ptr, velocity2Ptr);

        std::cout << "\nSaving frame\n";
        DrawFrame<<<numBlocks, threadsPerBlock>>>(theta1Ptr, theta2Ptr, imagePtr);
        hipMemcpy(image, imagePtr, imgSize, hipMemcpyDeviceToHost);

        fpng::fpng_encode_image_to_file(("frames/frame-" + std::to_string(i) + ".png").c_str(), image, SIZE, SIZE, 4);
    }

    hipFree(theta1Ptr);
    hipFree(theta2Ptr);
    hipFree(velocity1Ptr);
    hipFree(velocity2Ptr);
    hipFree(imagePtr);

    return 0;
}


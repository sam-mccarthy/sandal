#include "hip/hip_runtime.h"
#include <format>
#include <SDL.h>
#include <iostream>
#include "mandelbrot.cuh"
#include "fpng.cuh"

__global__ void CalculateFrame(int width, int height, unsigned char* image, int maxIter, unsigned long long scaleNum, unsigned long long scaleDen, unsigned long long panXnum, unsigned long long panXden, float panY, float* reference){
    unsigned int px = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int py = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = py * width + px;

    if(index > width * height) return;

    float dx0 = ((float)px / width * 4 - 2) / scale + panX;
    float dy0 = ((float)py / height * 4 - 2) / scale + panY;

    float x = 0;
    float y = 0;
    float x2 = 0;
    float y2 = 0;

    int iter = 0;
    while(x2 + y2 <= 4 && iter < maxIter){
        y = (x + x) * y + dy0;
        x = x2 - y2 + dx0;

        x2 = x * x;
        y2 = y * y;

        iter++;
    }

    float color = iter;
    if(iter < maxIter){
        color++;

        float log_zn = log2f(sqrt(x2 + y2));
        color -= log2f(log_zn);
    }

    unsigned int imageIndex = index * 4;

    image[imageIndex + 0] = (unsigned char)(sin(color) * 127 + 128);
    image[imageIndex + 1] = (unsigned char)(cos(color) * 127 + 128);
    image[imageIndex + 2] = (unsigned char)(tan(color) * 127 + 128);
    image[imageIndex + 3] = 255;
}

Mandelbrot::Mandelbrot(int w, int h, int iter) {
    width = w;
    height = h;
    maxIter = iter;

    blockSize = dim3(32, 32);
    gridSize = dim3((int)ceil((float)width / blockSize.x), (int)ceil((float)height / blockSize.y));

    image = new unsigned char[w * h * 4];
    hipMalloc((void **)&imageptr, w * h * 4);
    hipMalloc((void **)&referenceptr, maxIter * 2);

    reference = new float[iter * 2];
}

void Mandelbrot::CalculateReference(double x0, double y0, double scale) {
    x0 -= 2 / scale;
    y0 -= 2 / scale;

    double x = 0;
    double y = 0;
    double x2 = 0;
    double y2 = 0;

    int iter = 0;
    while(x2 + y2 <= 4 && iter < maxIter)
    {
        reference[iter * 2] = (float)x;
        reference[iter * 2 + 1] = (float)y;

        y = (x + x) * y + y0;
        x = x2 - y2 + x0;

        x2 = x * x;
        y2 = y * y;

        iter++;
    }

    hipMemcpy(referenceptr, reference, iter * 2, hipMemcpyHostToDevice);
}

void Mandelbrot::RenderFrame(double scale, double panX, double panY){
    CalculateReference(panX, panY, scale);
    CalculateFrame<<<gridSize, blockSize>>>(width, height, imageptr, maxIter, scale, panX, panY, referenceptr);
    hipMemcpy(image, imageptr, width * height * 4, hipMemcpyDeviceToHost);

    filename = std::format("{}{}{}{}{}.png", width, height, scale, panX, panY);
}

void Mandelbrot::RenderSDL(){
    if(SDL_Init(SDL_INIT_VIDEO) < 0){
        std::cout << "SDL Init Failed.";
        return;
    }

    SDL_Window* window;
    SDL_Renderer* renderer;
    SDL_CreateWindowAndRenderer(width, height, 0, &window, &renderer);

    if(window == nullptr || renderer == nullptr){
        std::cout << "Window / renderer creation error.";
        return;
    }

    SDL_Surface* surface = SDL_GetWindowSurface(window);
    SDL_Texture* mandelTexture = SDL_CreateTexture(renderer, SDL_PIXELFORMAT_RGBA32, SDL_TEXTUREACCESS_STREAMING, width, height);

    SDLEventLoop(mandelTexture, renderer);

    SDL_FreeSurface(surface);
    SDL_DestroyWindow(window);
    SDL_Quit();
}

void Mandelbrot::SDLEventLoop(SDL_Texture* texture, SDL_Renderer* renderer){
    bool quit = false;

    double scale = 1;

    double panX = 0;
    double panY = 0;

    bool change = true;
    bool mousePressed = false;
    SDL_Event event;
    while(!quit){
        if(SDL_PollEvent(&event)) {
            switch (event.type) {
                case SDL_KEYDOWN:
                    if (event.key.keysym.sym == SDLK_q) {
                        scale *= 0.9;
                        change = true;
                    } else if (event.key.keysym.sym == SDLK_e) {
                        scale *= 1.1;
                        change = true;
                    }
                    break;
                case SDL_MOUSEBUTTONDOWN:
                    mousePressed = true;
                    break;
                case SDL_MOUSEBUTTONUP:
                    mousePressed = false;
                    break;
                case SDL_MOUSEMOTION:
                    if (mousePressed) {
                        panX -= (float)event.motion.xrel * 4.0f / width / scale;
                        panY -= (float)event.motion.yrel * 4.0f / height / scale;
                        std::cout << panX << std::endl;
                        std::cout << panY << std::endl;
                        change = true;
                    }
                    break;
                case SDL_QUIT:
                    quit = true;
                    break;
                default:
                    break;
            }
        }

        if(change)
            UpdateSDL(texture, renderer, scale, panX, panY);

        change = false;
    }
}

void Mandelbrot::UpdateSDL(SDL_Texture* texture, SDL_Renderer* renderer, double scale, double panX, double panY){
    int pitch;
    void* pixels;

    RenderFrame(scale, panX, panY);

    SDL_LockTexture(texture, nullptr, &pixels, &pitch);
    memcpy(pixels, image, width * height * 4);
    SDL_UnlockTexture(texture);
    SDL_RenderCopy(renderer, texture, nullptr, nullptr);
    SDL_RenderPresent(renderer);
}

void Mandelbrot::SaveFrame(){
    fpng::fpng_encode_image_to_file(filename.c_str(), image, width, height, 4);
}

Mandelbrot::~Mandelbrot(){
    hipFree(imageptr);

    delete[] image;
    delete[] reference;
}